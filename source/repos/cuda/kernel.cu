#define _CRT_SECURE_NO_WARNINGS

#include <stdio.h>
#include <hip/hip_runtime.h>


//2015253039 ������

__global__ void helloWorld(char* str) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	str[idx] += idx;
}

int main(int argc, char** argv)
{
	int i;
	char strin[12] = "Hello";
	char str[] = "Hello World!";
	printf("%s", strin);

	for (i = 0; i < 12; i++)
	{
		str[i] -= i;
	}

	printf("%s\n", str);

	char* d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);

	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

	dim3 dimBlock(2);
	dim3 dimThread(6);

	helloWorld<<< dimBlock, dimThread >>>(d_str);

	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

	hipFree(d_str);

	printf("%s\n", str);
	return 0;
}